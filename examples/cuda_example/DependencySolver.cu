#include "hip/hip_runtime.h"
#include "Manager.h"
#include <unistd.h>
#include <assert.h>
#include <stdio.h>

/********** Functions to Run *****************/
void printer0(){
	std::cout << "Hello from 0\n";
	return;
}

int printer1(){
	std::this_thread::sleep_for(std::chrono::microseconds(500));
	std::cout << "Hello from 1\n";
	return 1;
}

int printer2(){
	std::cout << "Hello from 2\n";
	return 2;
}

void printer3(){
	std::cout << "Hello from 3\n";
	return;
}

#define COLUMNS 3
#define ROWS 2

__global__ void add(int *a, int *b, int *c)
{
 int x = blockIdx.x;
 int y = blockIdx.y;
 int i = (COLUMNS*y) + x;
 c[i] = a[i] + b[i];
}

void cuda_func(){

	int a[ROWS][COLUMNS], b[ROWS][COLUMNS], c[ROWS][COLUMNS];
	int *dev_a, *dev_b, *dev_c;
	
	hipMalloc((void **) &dev_a, ROWS*COLUMNS*sizeof(int));
	hipMalloc((void **) &dev_b, ROWS*COLUMNS*sizeof(int));
	hipMalloc((void **) &dev_c, ROWS*COLUMNS*sizeof(int));
	
	for (int y = 0; y < ROWS; y++){// Fill Arrays
		for (int x = 0; x < COLUMNS; x++)
		{
			a[y][x] = x;
			b[y][x] = y;
		}
	}

	hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int),
	hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, ROWS*COLUMNS*sizeof(int),
	hipMemcpyHostToDevice);
	
	dim3 grid(COLUMNS,ROWS);
	add<<<grid,1>>>(dev_a, dev_b, dev_c);
	
	hipMemcpy(c, dev_c, ROWS*COLUMNS*sizeof(int),
	hipMemcpyDeviceToHost);
	
	for (int y = 0; y < ROWS; y++) // Output Arrays
	{
		for (int x = 0; x < COLUMNS; x++)
		{
			printf("[%d][%d]=%d ",y,x,c[y][x]);
		}
		printf("\n");
	}
}
/*********************************************/

int main()
{
	Manager m;

	auto& node0 = m.append_node(0, printer0);
	auto& node1 = m.append_node(1, printer1);
	auto& node2 = m.append_node(2, printer2);
	auto& node3 = m.append_node(3, printer3);
	auto& node4 = m.append_node(4, cuda_func);

	// Dependeny Declaration
	// Node >> Node_1 implies Node is parent of Node_1
	// Node << Node_1 implies Node is child of Node_1

	node0 >> node1 >> node2;

	node3 << node1 << node2;

	node4 << node3;

	m.execute();

	// Expected Order of Execution
	std::vector<int> expected_order = {0, 2, 1, 3, 4};

	assert(expected_order == m.execution_order() && "Graph didn't execute in expected order!!");
	
	return 0;
}
